
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime_api.h>

__global__ void parallel_for(const int n, double* dax, double* dbx,
                             const double dt) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if (tid < n) {
        dax[tid] = dax[tid] + dbx[tid]*dt;
    }
}

int main()
{
  const int Nl = 1000;
  const double dt=0.001;
  const int N = 1000000;
  int blockSize = 64;
  int numBlocks = (N + blockSize -1) / blockSize;

  double* dax;
  double* dbx;

  hipMalloc((void**)&dax, sizeof(double)*N);
  hipMalloc((void**)&dbx, sizeof(double)*N);

  //warm up
  for(int j=0; j<100; j++)
  {
    parallel_for<<<numBlocks, blockSize>>>(N, dax, dbx, dt);
  }

  typedef std::chrono::high_resolution_clock Time;
  typedef std::chrono::duration<float> fsec;

  hipDeviceSynchronize();
  auto start_clock = Time::now();

  hipProfilerStart();

  for(int j=0; j<Nl; j++)
  {
    parallel_for<<<numBlocks, blockSize>>>(N, dax, dbx, dt);
  }
  
  hipDeviceSynchronize();
  
  hipProfilerStop();

  auto finish_clock = Time::now();
  fsec fs = finish_clock - start_clock;
  std::cout << "time taken for cuda parallel for (msecs):" << fs.count()*1e3/Nl << std::endl;

  hipFree(dax);
  hipFree(dbx);

  return 0; 
}
